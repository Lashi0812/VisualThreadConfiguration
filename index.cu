#include "hip/hip_runtime.h"
#include <iostream>

__forceinline__ __device__ unsigned getLaneId()
{
    unsigned ret; 
    asm volatile ("mov.u32 %0, %laneid;" : "=r"(ret));
    return ret;
}

__forceinline__ __device__ unsigned getWarpId()
{
    // this is not equal to threadIdx.x / 32
    unsigned ret; 
    asm volatile ("mov.u32 %0, %warpid;" : "=r"(ret));
    return ret;
}

__global__  void  index()
{
    auto warp = getWarpId();
    auto lane = getLaneId();

    printf("%d ,%d ,%d ,%d ,%d ,%d ,%d ,%d, %d\n",blockIdx.y,blockIdx.x,gridDim.y,gridDim.x,warp,threadIdx.y,threadIdx.x,blockDim.y,blockDim.x);
}

int main(int argc,char *argv[])
{
    if (argc != 5)
    {   
        printf("Usage index [grid.x] [grid.y] [block.x] [block.y]");
        return 0;
    }
    unsigned int grid_x = std::strtoul(argv[1], nullptr, 10);
    unsigned int grid_y = std::strtoul(argv[2], nullptr, 10);
    unsigned int block_x = std::strtoul(argv[3], nullptr, 10);
    unsigned int block_y = std::strtoul(argv[4], nullptr, 10);

    index<<<{grid_x,grid_y},{block_x,block_y}>>>();
    hipDeviceReset();
}